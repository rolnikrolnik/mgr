#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2013 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 *
 */

/*
 * This sample implements a conjugate graident solver on GPU
 * using CUBLAS and CUSPARSE
 *
 */

// includes, system
#include <stdlib.h>
#include <stdio.h>
#include <string.h>

/* Using updated (v2) interfaces to cublas and cusparse */
#include <hip/hip_runtime.h>
#include <hipsparse.h>
#include <hipblas.h>

// Utilities and system includes
#include <helper_functions.h>  // helper for shared functions common to CUDA SDK samples
#include <hip/hip_runtime_api.h>       // helper function CUDA error checking and intialization

const char *sSDKname     = "conjugateGradient";

double mclock(){
     struct timeval tp;

     double sec,usec;
     gettimeofday( &tp, NULL );
     sec    = double( tp.tv_sec );
     usec   = double( tp.tv_usec )/1E6;
     return sec + usec;
}


#define dot_BS     32
#define kernel_BS  32

/* genTridiag: generate a random tridiagonal symmetric matrix */
void genTridiag(int *I, int *J, float *val, int N, int nz)
{
    double RAND_MAXi = 1e6;
    double val_r     = 12.345 * 1e5;
    
    I[0] = 0, J[0] = 0, J[1] = 1;
    val[0] = (float)val_r/RAND_MAXi + 10.0f;
    val[1] = (float)val_r/RAND_MAXi;
    int start;

    for (int i = 1; i < N; i++)
    {
        if (i > 1)
        {
            I[i] = I[i-1]+3;
        }
        else
        {
            I[1] = 2;
        }

        start = (i-1)*3 + 2;
        J[start] = i - 1;
        J[start+1] = i;

        if (i < N-1)
        {
            J[start+2] = i + 1;
        }

        val[start] = val[start-1];
        val[start+1] = (float)val_r/RAND_MAXi + 10.0f;

        if (i < N-1)
        {
            val[start+2] = (float)val_r/RAND_MAXi;
        }
    }

    I[N] = nz;
}


void cgs_basic(int argc, char **argv, int N, int M){

    //int M = 0, N = 0, 
    int nz = 0, *I = NULL, *J = NULL;
    float *val = NULL;
    const float tol = 1e-10f;
    const int max_iter = 1000;
    float *x;
    float *rhs;
    float a, b, na, r0, r1;
    int *d_col, *d_row;
    float *d_val, *d_x, dot;
    float *d_r, *d_p, *d_Ax;
    int k;
    float alpha, beta, alpham1;

    // This will pick the best possible CUDA capable device
    hipDeviceProp_t deviceProp;
    int devID = findCudaDevice(argc, (const char **)argv);

    if (devID < 0)
    {
        printf("exiting...\n");
        exit(EXIT_SUCCESS);
    }

    checkCudaErrors(hipGetDeviceProperties(&deviceProp, devID));

    // Statistics about the GPU device
    printf("> GPU device has %d Multi-Processors, SM %d.%d compute capabilities\n\n",
           deviceProp.multiProcessorCount, deviceProp.major, deviceProp.minor);

    int version = (deviceProp.major * 0x10 + deviceProp.minor);

    if (version < 0x11)
    {
        printf("%s: requires a minimum CUDA compute 1.1 capability\n", sSDKname);
        hipDeviceReset();
        exit(EXIT_SUCCESS);
    }

    /* Generate a random tridiagonal symmetric matrix in CSR format */
    //M = N = 32*64;//10; //1048576;
    printf("M = %d, N = %d\n", M, N);
    nz = (N-2)*3 + 4;
    I = (int *)malloc(sizeof(int)*(N+1));
    J = (int *)malloc(sizeof(int)*nz);
    val = (float *)malloc(sizeof(float)*nz);
    genTridiag(I, J, val, N, nz);

    /*
    for (int i = 0; i < nz; i++){
        printf("%d\t", J[i]);
    }
    printf("\n");
    for (int i = 0; i < nz; i++){
        printf("%2f\t", val[i]);
    }
    */

    x = (float *)malloc(sizeof(float)*N);
    rhs = (float *)malloc(sizeof(float)*N);

    for (int i = 0; i < N; i++)
    {
        rhs[i] = 1.0;
        x[i] = 0.0;
    }

    /* Get handle to the CUBLAS context */
    hipblasHandle_t cublasHandle = 0;
    hipblasStatus_t hipblasStatus_t;
    hipblasStatus_t = hipblasCreate(&cublasHandle);

    checkCudaErrors(hipblasStatus_t);

    /* Get handle to the CUSPARSE context */
    hipsparseHandle_t cusparseHandle = 0;
    hipsparseStatus_t cusparseStatus;
    cusparseStatus = hipsparseCreate(&cusparseHandle);

    checkCudaErrors(cusparseStatus);

    hipsparseMatDescr_t descr = 0;
    cusparseStatus = hipsparseCreateMatDescr(&descr);

    checkCudaErrors(cusparseStatus);

    hipsparseSetMatType(descr,HIPSPARSE_MATRIX_TYPE_GENERAL);
    hipsparseSetMatIndexBase(descr,HIPSPARSE_INDEX_BASE_ZERO);

    checkCudaErrors(hipMalloc((void **)&d_col, nz*sizeof(int)));
    checkCudaErrors(hipMalloc((void **)&d_row, (N+1)*sizeof(int)));
    checkCudaErrors(hipMalloc((void **)&d_val, nz*sizeof(float)));
    checkCudaErrors(hipMalloc((void **)&d_x, N*sizeof(float)));
    checkCudaErrors(hipMalloc((void **)&d_r, N*sizeof(float)));
    checkCudaErrors(hipMalloc((void **)&d_p, N*sizeof(float)));
    checkCudaErrors(hipMalloc((void **)&d_Ax, N*sizeof(float)));

    hipMemcpy(d_col, J, nz*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_row, I, (N+1)*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_val, val, nz*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_x, x, N*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_r, rhs, N*sizeof(float), hipMemcpyHostToDevice);

    alpha = 1.0;
    alpham1 = -1.0;
    beta = 0.0;
    r0 = 0.;


    double t_start = mclock();
    hipsparseScsrmv(cusparseHandle,HIPSPARSE_OPERATION_NON_TRANSPOSE, N, N, nz, &alpha, descr, d_val, d_row, d_col, d_x, &beta, d_Ax);

    hipblasSaxpy(cublasHandle, N, &alpham1, d_Ax, 1, d_r, 1);                                // PODMIEN FUNCKJE (I)
    hipblasStatus_t = hipblasSdot(cublasHandle, N, d_r, 1, d_r, 1, &r1);                        // PODMIEN FUNCKJE (II)

    k = 1;

    while (r1 > tol*tol && k <= max_iter)
    {
        if (k > 1)
        {
            b = r1 / r0;
            hipblasStatus_t = hipblasSscal(cublasHandle, N, &b, d_p, 1);                        // PODMIEN FUNCKJE (I)
            hipblasStatus_t = hipblasSaxpy(cublasHandle, N, &alpha, d_r, 1, d_p, 1);            // PODMIEN FUNCKJE (I)
        }
        else
        {
            hipblasStatus_t = hipblasScopy(cublasHandle, N, d_r, 1, d_p, 1);                    // PODMIEN FUNCKJE (I)
        }

        hipsparseScsrmv(cusparseHandle, HIPSPARSE_OPERATION_NON_TRANSPOSE, N, N, nz, &alpha, descr, d_val, d_row, d_col, d_p, &beta, d_Ax); // PODMIEN FUNCKJE (III)
        hipblasStatus_t = hipblasSdot(cublasHandle, N, d_p, 1, d_Ax, 1, &dot);                  // PODMIEN FUNCKJE (II)
        a = r1 / dot;

        hipblasStatus_t = hipblasSaxpy(cublasHandle, N, &a, d_p, 1, d_x, 1);                    // PODMIEN FUNCKJE (I)
        na = -a;
        hipblasStatus_t = hipblasSaxpy(cublasHandle, N, &na, d_Ax, 1, d_r, 1);                  // PODMIEN FUNCKJE (I)

        r0 = r1;
        hipblasStatus_t = hipblasSdot(cublasHandle, N, d_r, 1, d_r, 1, &r1);                    // PODMIEN FUNCKJE (II)
        hipDeviceSynchronize();
        printf("iteration = %3d, residual = %e\n", k, sqrt(r1));
        k++;
    }
    printf("TIME OF CGS_BASIC = %f\n", mclock() - t_start);

    hipMemcpy(x, d_x, N*sizeof(float), hipMemcpyDeviceToHost);

    float rsum, diff, err = 0.0;

    for (int i = 0; i < N; i++)
    {
        rsum = 0.0;

        for (int j = I[i]; j < I[i+1]; j++)
        {
            rsum += val[j]*x[J[j]];
        }

        diff = fabs(rsum - rhs[i]);

        if (diff > err)
        {
            err = diff;
        }
    }

    hipsparseDestroy(cusparseHandle);
    hipblasDestroy(cublasHandle);

    free(I);
    free(J);
    free(val);
    free(x);
    free(rhs);
    hipFree(d_col);
    hipFree(d_row);
    hipFree(d_val);
    hipFree(d_x);
    hipFree(d_r);
    hipFree(d_p);
    hipFree(d_Ax);

    hipDeviceReset();

    printf("Test Summary:  Error amount = %e\n", err);
    //exit((k <= max_iter) ? 0 : 1);


}

__global__ void axpy(float *d_p, float *d_r, float alpha, int numElements){
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < numElements)
    {
        d_p[i] =  d_r[i]*alpha + d_p[i];
    }
}

__global__ void scal(float *d_p, float alpha, int numElements){
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < numElements)
    {
        d_p[i] =  d_p[i]*alpha;
    }
}

__global__ void cpy(float *d_p, float *d_r, int numElements){
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < numElements)
    {
        d_p[i] =  d_r[i];
    }
}


__global__ void csrmv(float *d_Ax, int *d_col, int *d_row, float *d_val, float *d_x, int numElements){
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if(i < numElements){
        float sub = 0.0;
        int j;
        for (j = d_row[i]; j < d_row[i+1]; j++){
            sub += d_val[j] * d_x[d_col[j]];

        }
        d_Ax[i] = sub;
    }
}

void cgs_TODO(int argc, char **argv, int N, int M){

    //int M = 0, N = 0, 
    int nz = 0, *I = NULL, *J = NULL;
    float *val = NULL;
    const float tol = 1e-10f;
    const int max_iter = 1000;
    float *x;
    float *rhs;
    float a, b, na, r0, r1;
    int *d_col, *d_row;
    float *d_val, *d_x, dot;
    float *d_r, *d_p, *d_Ax;
    int k;
    float alpha, beta, alpham1;

    // This will pick the best possible CUDA capable device
    hipDeviceProp_t deviceProp;
    int devID = findCudaDevice(argc, (const char **)argv);

    if (devID < 0)
    {
        printf("exiting...\n");
        exit(EXIT_SUCCESS);
    }

    checkCudaErrors(hipGetDeviceProperties(&deviceProp, devID));

    // Statistics about the GPU device
    printf("> GPU device has %d Multi-Processors, SM %d.%d compute capabilities\n\n",
           deviceProp.multiProcessorCount, deviceProp.major, deviceProp.minor);

    int version = (deviceProp.major * 0x10 + deviceProp.minor);

    if (version < 0x11)
    {
        printf("%s: requires a minimum CUDA compute 1.1 capability\n", sSDKname);
        hipDeviceReset();
        exit(EXIT_SUCCESS);
    }

    /* Generate a random tridiagonal symmetric matrix in CSR format */
    //M = N = 32*64;//10; //1048576;
    printf("M = %d, N = %d\n", M, N);
    nz = (N-2)*3 + 4;
    I = (int *)malloc(sizeof(int)*(N+1));
    J = (int *)malloc(sizeof(int)*nz);
    val = (float *)malloc(sizeof(float)*nz);
    genTridiag(I, J, val, N, nz);

    /*
    for (int i = 0; i < nz; i++){
        printf("%d\t", J[i]);
    }
    printf("\n");
    for (int i = 0; i < nz; i++){
        printf("%2f\t", val[i]);
    }
    */

    x = (float *)malloc(sizeof(float)*N);
    rhs = (float *)malloc(sizeof(float)*N);

    for (int i = 0; i < N; i++)
    {
        rhs[i] = 1.0;
        x[i] = 0.0;
    }

    /* Get handle to the CUBLAS context */
    hipblasHandle_t cublasHandle = 0;
    hipblasStatus_t hipblasStatus_t;
    hipblasStatus_t = hipblasCreate(&cublasHandle);

    checkCudaErrors(hipblasStatus_t);

    /* Get handle to the CUSPARSE context */
    hipsparseHandle_t cusparseHandle = 0;
    hipsparseStatus_t cusparseStatus;
    cusparseStatus = hipsparseCreate(&cusparseHandle);

    checkCudaErrors(cusparseStatus);

    hipsparseMatDescr_t descr = 0;
    cusparseStatus = hipsparseCreateMatDescr(&descr);

    checkCudaErrors(cusparseStatus);

    hipsparseSetMatType(descr,HIPSPARSE_MATRIX_TYPE_GENERAL);
    hipsparseSetMatIndexBase(descr,HIPSPARSE_INDEX_BASE_ZERO);

    checkCudaErrors(hipMalloc((void **)&d_col, nz*sizeof(int)));
    checkCudaErrors(hipMalloc((void **)&d_row, (N+1)*sizeof(int)));
    checkCudaErrors(hipMalloc((void **)&d_val, nz*sizeof(float)));
    checkCudaErrors(hipMalloc((void **)&d_x, N*sizeof(float)));
    checkCudaErrors(hipMalloc((void **)&d_r, N*sizeof(float)));
    checkCudaErrors(hipMalloc((void **)&d_p, N*sizeof(float)));
    checkCudaErrors(hipMalloc((void **)&d_Ax, N*sizeof(float)));

    hipMemcpy(d_col, J, nz*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_row, I, (N+1)*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_val, val, nz*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_x, x, N*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_r, rhs, N*sizeof(float), hipMemcpyHostToDevice);

    alpha = 1.0;
    alpham1 = -1.0;
    beta = 0.0;
    r0 = 0.;

    int threadsPerBlock = 256;
    int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;

    // sparse matrix vector product: d_Ax = A * d_x
    //hipsparseScsrmv(cusparseHandle,HIPSPARSE_OPERATION_NON_TRANSPOSE, N, N, nz, &alpha, descr, d_val, d_row, d_col, d_x, &beta, d_Ax);  // PODMIEN FUNCKJE (ZADANIE-I)
    csrmv<<<blocksPerGrid, threadsPerBlock>>>(d_Ax, d_col, d_row, d_val, d_x, N);


    //azpy: d_r = d_r + alpham1 * d_Ax
    //hipblasSaxpy(cublasHandle, N, &alpham1, d_Ax, 1, d_r, 1);
   
    axpy<<<blocksPerGrid, threadsPerBlock>>>(d_r, d_Ax, alpham1, N);        			    // PODMIEN FUNCKJE (ZADANIE-I)
    //dot:  r1 = d_r * d_r
    hipblasStatus_t = hipblasSdot(cublasHandle, N, d_r, 1, d_r, 1, &r1);                        // PODMIEN FUNCKJE (ZADANIE-III)

    k = 1;

    while (r1 > tol*tol && k <= max_iter)
    {
        if (k > 1)
        {
            b = r1 / r0;
	    //scal: d_p = b * d_p
            //hipblasStatus_t = hipblasSscal(cublasHandle, N, &b, d_p, 1);
            scal<<<blocksPerGrid, threadsPerBlock>>>(d_p, b, N);                         // PODMIEN FUNCKJE (ZADANIE-I)
	    //axpy:  d_p = d_p + alpha * d_r
            //hipblasStatus_t = hipblasSaxpy(cublasHandle, N, &alpha, d_r, 1, d_p, 1);
            axpy<<<blocksPerGrid, threadsPerBlock>>>(d_p, d_r, alpha, N);               // PODMIEN FUNCKJE (ZADANIE-I)
        }
        else
        {
            //cpy: d_p = d_r
            //hipblasStatus_t = hipblasScopy(cublasHandle, N, d_r, 1, d_p, 1);
            cpy<<<blocksPerGrid, threadsPerBlock>>>(d_p, d_r, N);                   // PODMIEN FUNCKJE (ZADANIE-I)
        }

        //sparse matrix-vector product: d_Ax = A * d_p
        //hipsparseScsrmv(cusparseHandle, HIPSPARSE_OPERATION_NON_TRANSPOSE, N, N, nz, &alpha, descr, d_val, d_row, d_col, d_p, &beta, d_Ax); // PODMIEN FUNCKJE (ZADANIE-II)
        csrmv<<<blocksPerGrid, threadsPerBlock>>>(d_Ax, d_col, d_row, d_val, d_p, N);
        hipblasStatus_t = hipblasSdot(cublasHandle, N, d_p, 1, d_Ax, 1, &dot);                  // PODMIEN FUNCKJE (ZADANIE-III)
        a = r1 / dot;

        //axpy: d_x = d_x + a*d_p
        //hipblasStatus_t = hipblasSaxpy(cublasHandle, N, &a, d_p, 1, d_x, 1);
        axpy<<<blocksPerGrid, threadsPerBlock>>>(d_x, d_p, a, N);                       // PODMIEN FUNCKJE (ZADANIE-I)
        na = -a;
	 
        //axpy:  d_r = d_r + na * d_Ax
        //hipblasStatus_t = hipblasSaxpy(cublasHandle, N, &na, d_Ax, 1, d_r, 1);                  // PODMIEN FUNCKJE (ZADANIE-I)
        axpy<<<blocksPerGrid, threadsPerBlock>>>(d_r, d_Ax, na, N);
        r0 = r1;
	
        //dot: r1 = d_r * d_r
        hipblasStatus_t = hipblasSdot(cublasHandle, N, d_r, 1, d_r, 1, &r1);                    // PODMIEN FUNCKJE (ZADANIE-III)
        hipDeviceSynchronize();
        printf("iteration = %3d, residual = %e\n", k, sqrt(r1));
        k++;
    }

    hipMemcpy(x, d_x, N*sizeof(float), hipMemcpyDeviceToHost);

    float rsum, diff, err = 0.0;

    for (int i = 0; i < N; i++)
    {
        rsum = 0.0;

        for (int j = I[i]; j < I[i+1]; j++)
        {
            rsum += val[j]*x[J[j]];
        }

        diff = fabs(rsum - rhs[i]);

        if (diff > err)
        {
            err = diff;
        }
    }

    hipsparseDestroy(cusparseHandle);
    hipblasDestroy(cublasHandle);

    free(I);
    free(J);
    free(val);
    free(x);
    free(rhs);
    hipFree(d_col);
    hipFree(d_row);
    hipFree(d_val);
    hipFree(d_x);
    hipFree(d_r);
    hipFree(d_p);
    hipFree(d_Ax);

    hipDeviceReset();

    printf("Test Summary:  Error amount = %e\n", err);
    //exit((k <= max_iter) ? 0 : 1);

}



int main(int argc, char **argv)
{
    //int N = 1e6;//1 << 20;
    //int N = 256 * (1<<10)  -10 ; //1e6;//1 << 20;
    int N = 1e5;
    int M = N; 
    
    cgs_basic(argc, argv, N, M);
    
    cgs_TODO(argc, argv, N, M);
}
